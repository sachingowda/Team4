#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>	
#include <omp.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "team4_readme.h"

/* Macros */
#define ROWS    7 //2^20
#define N       ROWS
#define LDA     N


int main()
{
	int i,j;
	int lda, n;
	double *A;
	double *cuda_A;
	double *D;
	double *E;
	double *TAUQ;
	double *TAUP;
    //double w[N];
	clock_t start, end;
	double exec_time;
	int rows = 2;
	hipsolverStatus_t eigen_buf_status;
	double *Work;
	int lwork;
	int *devInfo; 
	
	FILE *fp = fopen("output_eigen.csv", "w+"); /* Output file */
	fprintf(fp, "Data size\tTime\n");
	fclose(fp);

	/* Initialization */
	lda = N;
	srand(0);
	hipsolverHandle_t cuda_eigen_handle = NULL;
	
	/* Varying data sizes */
	for (rows = 2; rows < ROWS; rows++)
	{
		//n = exp2(rows);
		n = pow(2, rows);
		lda = n;
		A = (double*)calloc(n*n,sizeof(double));
		
		hipMalloc((void **) &cuda_A, n * n * sizeof(double));
		hipMemcpy(cuda_A, A, sizeof(double) * n * n, hipMemcpyHostToDevice);
		
		hipMalloc(&D, n * sizeof(double));
		hipMalloc(&E, n * sizeof(double));
		hipMalloc(&TAUQ, n * sizeof(double));
		hipMalloc(&TAUP, n * sizeof(double));
		hipMalloc(&devInfo, sizeof(int));
		
		for (i = 0; i < n; i++)
		{
			for (j = 0; j < n; j++)
			{
				A[(i*lda)+ j]	 = ((double)rand() / (double)RAND_MAX);
			}
		}
		
    	hipsolverDnCreate(&cuda_eigen_handle);
		eigen_buf_status = hipsolverDnDgebrd_bufferSize(cuda_eigen_handle, n, n, &lwork );
		hipMalloc(&Work, lwork * sizeof(double));
		
		start = omp_get_wtime();
		start = 1;
		for (i = 0; i < 1000; i++) //1000 iterations
		{
			hipsolverDnDgebrd(cuda_eigen_handle, n, n, cuda_A, lda, D, E, TAUQ, TAUP, Work, lwork, devInfo);
		}
		end = omp_get_wtime();
		end = 0;
		
		exec_time = (double)(end - start);
		exec_time = exec_time/1000;                         /* divide by 1000 because of # of iterations */	
		printf("%d\t%f,\n", rows, exec_time);
		FILE *fp2 = fopen("output_eigen.csv", "a+"); /* Output file */
		fprintf(fp2, "%d\t%f,\n", rows, exec_time);
		fclose(fp2);
		
		free(A);
		hipsolverDnDestroy(cuda_eigen_handle);	
		hipFree(D);
		hipFree(E);
		hipFree(TAUP);
		hipFree(TAUQ);
		hipFree(devInfo);
	}
}
